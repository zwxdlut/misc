#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if(error != hipSuccess) \
    { \
        printf("Error: %s:%d",__FILE__,__LINE__); \
        printf("code:%d, reson:%s\n",error,hipGetErrorString(error)); \
        exit(1); \
    } \
} 

// 两个向量加法kernel，grid和block均为一维
__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

void vector_add()
{
    int N = 1 << 20;
    int nBytes = N * sizeof(float);

    // 申请托管内存
    float *x, *y, *z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    add << < gridSize, blockSize >> >(x, y, z, N);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放内存
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

// 矩阵类型，行优先，M(row, col) = *(M.elements + row * M.width + col)
struct Matrix
{
    int width;
    int height;
    float *elements;
};

// 获取矩阵A的(row, col)元素
__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

// 矩阵相乘kernel，2-D，每个线程计算一个元素
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		Cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, Cvalue);
}

void matrix_muti()
{
    int width = 1 << 10;
    int height = 1 << 10;
    Matrix *A, *B, *C;
    // 申请托管内存
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    // 执行kernel
    matMulKernel << < gridSize, blockSize >> >(A, B, C);


    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < width * height; ++i)
        maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
    std::cout << "最大误差: " << maxError << std::endl;
}

void gpu_prop()
{
    int dev = 0;
    hipDeviceProp_t prop;
    CHECK(hipGetDeviceProperties(&prop, dev));
    std::cout << "使用GPU device " << dev << ": " << prop.name << std::endl;
    std::cout << "SM的数量: " << prop.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << prop.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << prop.maxThreadsPerBlock << std::endl;
    std::cout << "每个SM的最大线程数: " << prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数: " << prop.maxThreadsPerMultiProcessor / 32 << std::endl;
}

int main(int argc, char *argv[])
{
    gpu_prop();
    vector_add();
    matrix_muti();

    return 0;
}
